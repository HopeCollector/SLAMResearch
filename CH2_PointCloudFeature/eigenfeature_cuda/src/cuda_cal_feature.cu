#include <vector>
#include <cmath>
#include <algorithm>
#include <hip/hip_runtime.h>

// cuda 需要再其运算单元中放入进行运算的函数，我们用
// __global__ 进行指定，这个函数被称为 kernel
// https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#programming-model
__global__
void _cal_feature(float* e, float* features)
{
    switch (threadIdx.x)
    {
    case 0:
        features[0] = (e[0] - e[1]) / e[0];
        break;

    case 1:
        features[1] = (e[1] - e[2]) / e[0];
        break;

    case 2:
        features[2] = e[2] / e[0];
        break;

    case 3:
        features[3] = std::cbrt(e[0]*3.0f + e[1]*3.0f + e[2]*3.0f);
        break;

    case 4:
        features[4] = -e[0] * std::log(e[0]) - e[1] * std::log(e[1]) - e[2] * std::log(e[2]);
        break;

    case 5:
        features[5] = 3.0f * e[2];
        break;
    
    default:
        break;
    }
}

void cal_feature(float*e, float*features)
{
    // cuda 自己新定义的运算符 <<<blockNum，threadNumEachBlock, memSize>>>
    // https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#execution-configuration
    _cal_feature<<<1,6>>>(e,features);
}